#include "hip/hip_runtime.h"
#include "global_to_shared.cuh"

#ifdef TEST_WARP_MEMORY_TILE_GLOBAL_TO_SHARED

template<typename T>
struct load_store {
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<NW == 1 &&
        W*H<=64>;
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "shared_loadstore_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "shared_loadstore_gmem=half" :
                                                                                         "shared_loadstore_gmem=float";
    template<int H, int W, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        o_ref = i_ref; // overwrite the whole thing
    }
    template<int H, int W, int NW> __device__ static void device_func(const T *input, T *output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator<16> al((int*)&__shm[0]); 
        kittens::st<T, H, W> &shared_tile = al.allocate<kittens::st<T, H, W>>();
        kittens::load(shared_tile, input, W*16);
        kittens::store(output, shared_tile, W*16);
    }
};
template<typename T>
struct load_store_async {
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<NW == 1 &&
        W*H<=64>;
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "shared_loadstore_async_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "shared_loadstore_async_gmem=half" :
                                                                                         "shared_loadstore_async_gmem=float";
    template<int H, int W, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        o_ref = i_ref; // overwrite the whole thing
    }
    template<int H, int W, int NW> __device__ static void device_func(const T *input, T *output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator<16> al((int*)&__shm[0]); 

        auto block = cooperative_groups::this_thread_block();
        __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;
        if (threadIdx.x == 0) {init(&barrier, block.size());}
        block.sync();
        
        kittens::st<T, H, W> &shared_tile = al.allocate<kittens::st<T, H, W>>();

        block.sync();
        kittens::load_async(shared_tile, input, W*16, barrier);
        barrier.arrive_and_wait();

        kittens::store_async(output, shared_tile, W*16, barrier);
        barrier.arrive_and_wait();
    }
};

void warp::memory::tile::global_to_shared::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/memory/tile/global_to_shared tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;
    sweep_gmem_type_2d_warp<load_store, SIZE, SIZE>::run(results);
    sweep_gmem_type_2d_warp<load_store_async, SIZE, SIZE>::run(results);
}

#endif