#include "hip/hip_runtime.h"
#include "tma_multicast.cuh"
#include <hip/hip_cooperative_groups.h>

#ifdef TEST_WARP_MEMORY_TILE_TMA_MULTICAST

template<typename T>
struct test_load_multicast { // load with TMA, write out normally
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<NW == 1 && W*H*sizeof(dtype)*256<=kittens::MAX_SHARED_MEMORY-4096>; // S%4 ensures alignment
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "tma_multicast_load_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "tma_multicast_load_gmem=half" :
                                                                                         "tma_multicast_load_gmem=float";
    template<int H, int W, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        int SIZE_DIV_4 = i_ref.size()/4;
        for(int i = 0; i < SIZE_DIV_4; i++) {
            for(int j = 0; j < 4; j++) {
                o_ref[i+j*SIZE_DIV_4] = i_ref[i];
            }
        }
    }
    template<int H, int W, int NW>
    __device__ static void device_func(const dtype *input, dtype *output, CUtensorMap* tma_desc_input, CUtensorMap* tma_desc_output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_swizzle_allocator al((int*)&__shm[0]); 
        kittens::st<dtype, H, W> (&shared_tile) = al.allocate<kittens::st<dtype, H, W>>();
        auto cluster = cooperative_groups::this_cluster();
        int rank = cluster.block_rank();
        
        __shared__ kittens::barrier smem_barrier; 
        kittens::init_barrier(smem_barrier, 0, 1);
        // *************************************************************************************************
        // Doing it this way would also work, but I want to illustrate the use of the cluster::expect, too.
        // kittens::tma::expect<typeof(shared_tile)>(smem_barrier);
        // *************************************************************************************************
        cluster.sync(); // ensure everyone has initialized their barrier

        if(rank == 0 && threadIdx.x == 0) { // only one block issues the multicast load for everyone
            for(int j = 0; j < 4; j++) { // expect on the whole block
                kittens::tma::cluster::expect<typeof(shared_tile)>(smem_barrier, j);
            }
            kittens::tma::cluster::load_async(shared_tile, tma_desc_input, smem_barrier, 0, 0, 0b1111);
        }

        kittens::wait(smem_barrier, 0);
        kittens::store(output + rank*shared_tile.num_elements, shared_tile, W*kittens::TILE_DIM);
        cluster.sync();
    }
};

template<typename Ker, typename T, int H, int W, int NW, typename... args>
static __global__ __cluster_dims__(4, 1, 1) void tmamulti_global_wrapper_2d(const T *input, T *output, CUtensorMap* tma_desc_input, CUtensorMap* tma_desc_output) {
    Ker::template device_func<H, W, NW, args...>(input, output, tma_desc_input, tma_desc_output);
}
template<typename test, int H, int W, int NUM_WORKERS, typename... args>
struct tmamulti_wrapper_2d {
    using dtype = gmem_dtype<test>; // defaults to bf16 in global memory if the test doesn't specify.
    static void run(test_data& results) {
        test_info this_result;
        this_result.label = generate_test_name<H, W, NUM_WORKERS, args...>(test::test_identifier);
        if constexpr (test::template valid<H, W, NUM_WORKERS, args...>::value) {
            constexpr int SIZE = H*W*256 * 4; // 4 for additional TMA dimension
            // initialize
            dtype *d_i, *d_o;
            std::vector<float> i_ref(SIZE);
            std::vector<float> o_ref(SIZE);
            initialize(&d_i, &d_o, i_ref, o_ref);
            // initialize TMA descriptors
            CUtensorMap *i_desc = kittens::tma::allocate_and_create_tensor_map<kittens::st<dtype, H, W>>(d_i, 4);
            CUtensorMap *o_desc = kittens::tma::allocate_and_create_tensor_map<kittens::st<dtype, H, W>>(d_o, 4);
            // run kernel
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                tmamulti_global_wrapper_2d<test), dtype, H, W, NUM_WORKERS, args...>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                kittens::MAX_SHARED_MEMORY
            );
            tmamulti_global_wrapper_2d<test, dtype, H, W, NUM_WORKERS, args...><<<4, NUM_WORKERS*32, kittens::MAX_SHARED_MEMORY>>>(d_i, d_o, i_desc, o_desc);
            // fill in correct results on cpu
            test::template host_func<H, W, NUM_WORKERS, args...>(i_ref, o_ref);
            // check and cleanup
            this_result.result = validate(d_i, d_o, i_ref, o_ref, this_result.label, W*kittens::TILE_DIM);
            hipFree(i_desc);
            hipFree(o_desc);
        }
        else {
            this_result.result = test_result::INVALID;
        }
        results.push_back(this_result);
    }
};
template<typename test, int MAX_H=8, int MAX_W=8, int NUM_WORKERS=1, typename... args>
using tmamulti_sweep_size_2d = loop_h<tmamulti_wrapper_2d, test, MAX_H, MAX_W, NUM_WORKERS, MAX_H, args...>;
template<template<typename> typename test, int MAX_H=8, int MAX_W=8, int NUM_WORKERS=1, typename... args>
struct tmamulti_sweep_gmem_type_2d {
    static void run(test_data &results) {
        tmamulti_sweep_size_2d<test<float>, MAX_H, MAX_W, NUM_WORKERS, args...>::run(results);
        tmamulti_sweep_size_2d<test<kittens::bf16>, MAX_H, MAX_W, NUM_WORKERS, args...>::run(results);
        tmamulti_sweep_size_2d<test<kittens::half>, MAX_H, MAX_W, NUM_WORKERS, args...>::run(results);
    }
};
template<template<typename> typename test, int MAX_H=8, int MAX_W=8, typename... args> using tmamulti_sweep_gmem_type_2d_warp = tmamulti_sweep_gmem_type_2d<test, MAX_H, MAX_W, 1, args...>;

void warp::memory::tile::tma_multicast::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/memory/tile/tma_multicast tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    tmamulti_sweep_gmem_type_2d_warp<test_load_multicast, SIZE, SIZE>::run(results);
}

#endif