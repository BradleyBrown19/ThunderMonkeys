#include "hip/hip_runtime.h"
#include "conversions.cuh"

#ifdef TEST_GROUP_SHARED_TILE_CONVERSIONS

struct test_shared_copy {
    template<int H, int W, int NW> using valid = std::bool_constant<H%NW==0 && W*H<=64>; // this is group-level
    static inline const std::string test_identifier = "shared_copy";
    template<int H, int W, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        o_ref = i_ref; // overwrite the whole thing
    }
    template<int H, int W, int NW> __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output) {
        using G = kittens::group<NW>;
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator al((int*)&__shm[0]); 
        kittens::st_bf<H, W> &t1 = al.allocate<kittens::st_bf<H, W>>();
        kittens::st_bf<H, W> &t2 = al.allocate<kittens::st_bf<H, W>>();
        G::load(t2, input, W*16);
        __syncthreads();
        G::copy(t1, t2);
        __syncthreads();
        G::store(output, t1, W*16);
    }
};

void group::shared::tile::conversions::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/group/shared/conversions tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    sweep_size_2d<test_shared_copy, SIZE, SIZE, 2>::run(results);

    if constexpr (TEST_INTENSITY > 1) {

        sweep_size_2d<test_shared_copy, SIZE, SIZE, 4>::run(results);

        if constexpr (TEST_INTENSITY > 3) {

            sweep_size_2d<test_shared_copy, 12, 4, 12>::run(results);

        }
    }
}

#endif